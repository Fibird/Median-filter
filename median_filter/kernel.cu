#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#define N 33 * 1024
#define threadsPerBlock 256
#define blocksPerGrid (N + threadsPerBlock - 1) / threadsPerBlock
#define RADIUS 2
// Signal/image element type
typedef int element;
//   1D MEDIAN FILTER implementation
//     signal - input signal
//     result - output signal
//     N      - length of the signal


__global__ void _medianfilter(const element* signal, element* result)
{
	__shared__ element cache[threadsPerBlock + 2 * RADIUS];
	element window[5];
	int gindex = threadIdx.x + blockDim.x * blockIdx.x;
	int lindex = threadIdx.x + RADIUS;
	// Reads input elements into shared memory
	cache[lindex] = signal[gindex];
	if (threadIdx.x < RADIUS)
	{
		cache[lindex - RADIUS] = signal[gindex - RADIUS];
		cache[lindex + threadsPerBlock] = signal[gindex + threadsPerBlock];
	}
	__syncthreads();
	for (int j = 0; j < 2 * RADIUS + 1; ++j)
		window[j] = cache[threadIdx.x + j];
	// Orders elements (only half of them)
	for (int j = 0; j < RADIUS + 1; ++j)
	{
		// Finds position of minimum element
		int min = j;
		for (int k = j + 1; k < 2 * RADIUS + 1; ++k)
			if (window[k] < window[min])
				min = k;
		// Puts found minimum element in its place
		const element temp = window[j];
		window[j] = window[min];
		window[min] = temp;
	}
	// Gets result - the middle element
	result[gindex] = window[RADIUS];
}

//   1D MEDIAN FILTER wrapper
//     signal - input signal
//     result - output signal
//     N      - length of the signal
void medianfilter(element* signal, element* result)
{
	element *dev_extension, *dev_result;

	//   Check arguments
	if (!signal || N < 1)
		return;
	//   Treat special case N = 1
	if (N == 1)
	{
		if (result)
			result[0] = signal[0];
		return;
	}
	//   Allocate memory for signal extension
	element* extension = (element*)malloc((N + 2 * RADIUS) * sizeof(element));
	//   Check memory allocation
	if (!extension)
		return;
	//   Create signal extension
	hipMemcpy(extension + 2, signal, N * sizeof(element), hipMemcpyHostToHost);
	for (int i = 0; i < RADIUS; ++i)
	{
		extension[i] = signal[1 - i];
		extension[N + RADIUS + i] = signal[N - 1 - i];
	}

	hipMalloc((void**)&dev_extension, (N + 2 * RADIUS) * sizeof(int));
	hipMalloc((void**)&dev_result, N * sizeof(int));

	// Copies signal to device
	hipMemcpy(dev_extension, extension, (N + 2 * RADIUS) * sizeof(element), hipMemcpyHostToDevice);
	//   Call median filter implementation
	for (int i = 0; i < 10; ++i)
		_medianfilter<<<blocksPerGrid, threadsPerBlock>>>(dev_extension + RADIUS, dev_result);
	// Copies result to host
	hipMemcpy(result, dev_result, N * sizeof(element), hipMemcpyDeviceToHost);

	// Free memory
	free(extension);
	hipFree(dev_extension);
	hipFree(dev_result);
}

int main()
{
	int *Signal, *result;
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	FILE *fp;
	
	Signal = (int *)malloc(N * sizeof(int));
	result = (element *)malloc(N * sizeof(element));
	
	for (int i = 0; i < N; i++)
	{
		Signal[i] = i % 5 + 1;
	}
	hipEventRecord(start, 0);
	medianfilter(Signal, result);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%.3lf ms\n", elapsedTime);

	fp = fopen("result.txt", "w");
	if (fp == NULL)
		printf("OPEN FILE FAILS!\n");
	for (int i = 0; i < N; i ++)
		fprintf(fp, "%d ", result[i]);

	fclose(fp);
	return 0;
}