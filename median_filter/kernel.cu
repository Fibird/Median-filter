#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>
#include <opencv2\core.hpp>
#include <opencv2\highgui\highgui.hpp>

#define threadsPerBlock 256
#define RADIUS 2

// Signal/image element type
typedef unsigned char element;
//   1D MEDIAN FILTER implementation
//     signal - input signal
//     result - output signal
//     N      - length of the signal


__global__ void _medianfilter(const element* signal, element* result)
{
	__shared__ element cache[threadsPerBlock + 2 * RADIUS];
	element window[5];
	int gindex = threadIdx.x + blockDim.x * blockIdx.x;
	int lindex = threadIdx.x + RADIUS;
	// Reads input elements into shared memory
	cache[lindex] = signal[gindex];
	if (threadIdx.x < RADIUS)
	{
		cache[lindex - RADIUS] = signal[gindex - RADIUS];
		cache[lindex + threadsPerBlock] = signal[gindex + threadsPerBlock];
	}
	__syncthreads();
	for (int j = 0; j < 2 * RADIUS + 1; ++j)
		window[j] = cache[threadIdx.x + j];
	// Orders elements (only half of them)
	for (int j = 0; j < RADIUS + 1; ++j)
	{
		// Finds position of minimum element
		int min = j;
		for (int k = j + 1; k < 2 * RADIUS + 1; ++k)
			if (window[k] < window[min])
				min = k;
		// Puts found minimum element in its place
		const element temp = window[j];
		window[j] = window[min];
		window[min] = temp;
	}
	// Gets result - the middle element
	result[gindex] = window[RADIUS];
}

//   1D MEDIAN FILTER wrapper
//     signal - input signal
//     result - output signal
//     N      - length of the signal
void medianfilter(element* signal, element* result, int N)
{
	element *dev_extension, *dev_result;

	//   Check arguments
	if (!signal || N < 1)
		return;
	//   Treat special case N = 1
	if (N == 1)
	{
		if (result)
			result[0] = signal[0];
		return;
	}
	//   Allocate memory for signal extension
	element* extension = (element*)malloc((N + 2 * RADIUS) * sizeof(element));
	//   Check memory allocation
	if (!extension)
		return;
	//   Create signal extension
	hipMemcpy(extension + 2, signal, N * sizeof(element), hipMemcpyHostToHost);
	for (int i = 0; i < RADIUS; ++i)
	{
		extension[i] = signal[1 - i];
		extension[N + RADIUS + i] = signal[N - 1 - i];
	}

	hipMalloc((void**)&dev_extension, (N + 2 * RADIUS) * sizeof(int));
	hipMalloc((void**)&dev_result, N * sizeof(int));

	// Copies signal to device
	hipMemcpy(dev_extension, extension, (N + 2 * RADIUS) * sizeof(element), hipMemcpyHostToDevice);
	//   Call median filter implementation
	_medianfilter<<<blocksPerGrid, threadsPerBlock>>>(dev_extension + RADIUS, dev_result);
	// Copies result to host
	hipMemcpy(result, dev_result, N * sizeof(element), hipMemcpyDeviceToHost);

	// Free memory
	free(extension);
	hipFree(dev_extension);
	hipFree(dev_result);
}

int main()
{
	IplImage *ImgSrc = cvLoadImage("sample_corrupted.bmp", CV_LOAD_IMAGE_GRAYSCALE);
	IplImage *ImgReal = cvLoadImage("sample.bmp", CV_LOAD_IMAGE_GRAYSCALE);
	IplImage *ImgDst_CPU = cvCreateImage(cvGetSize(ImgSrc), IPL_DEPTH_8U, 1);
	int Size = ImgSrc->width * ImgSrc->height;
	unsigned char *pSrcData = (unsigned char*)(ImgSrc->imageData);
	unsigned char *pDstData = (unsigned char*)(ImgDst_CPU->imageData);
	int blocksPerGrid = (Size + threadsPerBlock - 1) / threadsPerBlock;

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	medianfilter<<<blocksPerGrid, threadsPerBlock>>>(pSrcData, pDstData, Size);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%lf.3 ms\n", elapsedTime);

	return 0;
}