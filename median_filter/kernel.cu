#include "hip/hip_runtime.h"

#include <stdio.h>
#include <memory.h>
#include <algorithm>

#define N 33 * 1024
#define threadsPerBlock 256
#define blocksPerGrid (N + threadsPerBlock - 1) / threadsPerBlock
#define RADIUS 2
// Signal/image element type
typedef int element;
//   1D MEDIAN FILTER implementation
//     signal - input signal
//     result - output signal
//     N      - length of the signal


__global__ void _medianfilter(const element* signal, element* result)
{
	element window[5];
	__shared__ element cache[threadsPerBlock + 2 * RADIUS];
	int gindex = threadIdx.x + blockDim.x * blockIdx.x;
	int lindex = threadIdx.x + RADIUS;
	// Reads input elements into shared memory
	cache[lindex] = signal[gindex];
	if (threadIdx.x < RADIUS)
	{
		cache[lindex - RADIUS] = signal[gindex - RADIUS];
		cache[lindex + threadsPerBlock] = signal[gindex + threadsPerBlock];
	}
	__syncthreads();
	for (int j = 0; j < 5; ++j)
		window[j] = cache[threadIdx.x  + j];
	// Orders elements (only half of them)
	for (int j = 0; j < 3; ++j)
	{
		// Finds position of minimum element
		int min = j;
		for (int k = j + 1; k < 5; ++k)
			if (window[k] < window[min])
				min = k;
		// Puts found minimum element in its place
		const element temp = window[j];
		window[j] = window[min];
		window[min] = temp;
	}
	// Gets result - the middle element
	result[gindex] = window[2];
}

//   1D MEDIAN FILTER wrapper
//     signal - input signal
//     result - output signal
//     N      - length of the signal
void medianfilter(element* signal, element* result)
{
	//   Check arguments
	if (!signal || N < 1)
		return;
	//   Treat special case N = 1
	if (N == 1)
	{
		if (result)
			result[0] = signal[0];
		return;
	}
	//   Allocate memory for signal extension
	element* extension = (element*)malloc((N + 2 * RADIUS) * sizeof(element));
	//   Check memory allocation
	if (!extension)
		return;
	//   Create signal extension
	hipMemcpy(extension + 2, signal, N * sizeof(element), hipMemcpyHostToHost);
	for (int i = 0; i < 2; ++i)
	{
		extension[i] = signal[1 - i];
		extension[N + 2 + i] = signal[N - 1 - i];
	}

	element *dev_extension, *dev_result;
	dev_extension = (element *)hipMalloc((void**)&dev_extension, (N + 2 * RADIUS) * sizeof(int));
	dev_result = (int *)hipMalloc((void**)&dev_result, N * sizeof(int));
	
	// Copies signal to device
	hipMemcpy(dev_extension, extension, (N + 2 * RADIUS) * sizeof(element), hipMemcpyHostToDevice);
	//   Call median filter implementation
	_medianfilter<<<blocksPerGrid, threadsPerBlock>>>(dev_extension, dev_result);
	// Copies result to host
	hipMemcpy(result, dev_result, N * sizeof(element), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i ++)
		printf("%d ", result[i]);
	// Free memory
	free(extension);
	hipFree(dev_extension);
	hipFree(dev_result);
}

int main()
{
	int *Signal, *result;
	Signal = (int *)malloc(N * sizeof(int));
	result = (element *)malloc(N * sizeof(element));
	//fill_n(Signal, N, 1);
	for (int i = 0; i < N; i++)
		Signal[i] = 1;
	medianfilter(Signal, result);
	
	return 0;
}