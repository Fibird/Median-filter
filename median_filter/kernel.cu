#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#define N 33 * 1024
#define threadsPerBlock 256
#define blocksPerGrid (N + threadsPerBlock - 1) / threadsPerBlock
#define RADIUS 2
// Signal/image element type
typedef int element;
//   1D MEDIAN FILTER implementation
//     signal - input signal
//     result - output signal
//     N      - length of the signal


__global__ void _medianfilter(const element* signal, element* result)
{
	__shared__ element cache[threadsPerBlock + 2 * RADIUS];
	element window[5];
	int gindex = threadIdx.x + blockDim.x * blockIdx.x;
	int lindex = threadIdx.x + RADIUS;
	// Reads input elements into shared memory
	cache[lindex] = signal[gindex];
	if (threadIdx.x < RADIUS)
	{
		cache[lindex - RADIUS] = signal[gindex - RADIUS];
		cache[lindex + threadsPerBlock] = signal[gindex + threadsPerBlock];
	}
	__syncthreads();
	for (int j = 0; j < 5; ++j)
		window[j] = cache[threadIdx.x  + j];
	// Orders elements (only half of them)
	for (int j = 0; j < 3; ++j)
	{
		// Finds position of minimum element
		int min = j;
		for (int k = j + 1; k < 5; ++k)
			if (window[k] < window[min])
				min = k;
		// Puts found minimum element in its place
		const element temp = window[j];
		window[j] = window[min];
		window[min] = temp;
	}
	// Gets result - the middle element
	result[gindex] = window[2];
}

//   1D MEDIAN FILTER wrapper
//     signal - input signal
//     result - output signal
//     N      - length of the signal
void medianfilter(element* signal, element* result)
{
	element *dev_extension, *dev_result;

	//   Check arguments
	if (!signal || N < 1)
		return;
	//   Treat special case N = 1
	if (N == 1)
	{
		if (result)
			result[0] = signal[0];
		return;
	}
	//   Allocate memory for signal extension
	element* extension = (element*)malloc((N + 2 * RADIUS) * sizeof(element));
	//   Check memory allocation
	if (!extension)
		return;
	//   Create signal extension
	hipMemcpy(extension + 2, signal, N * sizeof(element), hipMemcpyHostToHost);
	for (int i = 0; i < 2; ++i)
	{
		extension[i] = signal[1 - i];
		extension[N + 2 + i] = signal[N - 1 - i];
	}

	hipMalloc((void**)&dev_extension, (N + 2 * RADIUS) * sizeof(int));
	hipMalloc((void**)&dev_result, N * sizeof(int));

	// Copies signal to device
	hipMemcpy(dev_extension, extension, (N + 2 * RADIUS) * sizeof(element), hipMemcpyHostToDevice);
	//   Call median filter implementation
	_medianfilter<<<blocksPerGrid, threadsPerBlock>>>(dev_extension, dev_result);
	// Copies result to host
	hipMemcpy(result, dev_result, N * sizeof(element), hipMemcpyDeviceToHost);

	// Free memory
	free(extension);
	hipFree(dev_extension);
	hipFree(dev_result);
}

int main()
{
	int *Signal, *result;
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	FILE *fp;
	
	Signal = (int *)malloc(N * sizeof(int));
	result = (element *)malloc(N * sizeof(element));
	
	for (int i = 0; i < N; i++)
	{
		Signal[i] = i % 5 + 1;
	}
	hipEventRecord(start, 0);
	medianfilter(Signal, result);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%lf.3 ms\n", elapsedTime);

	fp = fopen("result.txt", "w");
	if (fp == NULL)
		printf("OPEN FILE FAILS!\n");
	for (int i = 0; i < N; i ++)
		fprintf(fp, "%d ", result[i]);

	return 0;
}